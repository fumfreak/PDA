#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


#define width = 10000;
#define height = 10000;



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void Reduce_noise(int *devPtr, int *pitch )
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	float x = ((devPtr + (j * pitch + i - 1) + (devPtr + (j * pitch + i + 1))/2;
	//devPtr + (j * pitch + i) = x;
	
	if (i == 0) {
		
	}

	if (i == width - 1) {

	}
	
	if (j == 0) {

	}

	if (j == height - 1) {

	}

	


    
}

int main()
{
    
	
	float *h_A[width][height];
	
	size_t pitch;
	float *devPtr;

	hipMallocPitch (&devPtr, &pitch, width * sizeof(float), height * sizeof(float));
	hipMemcpy2D(devPtr, pitch, h_A, pitch, width* sizeof(float), height* sizeof(float), hipMemcpyHostToDevice);

	dim3 threadsPerBlock = (width, height);
	dim3 blocksPerGrid = 1;

	Reduce_noise<<<blocksPerGrid, threadsPerBlock>>>(devPtr, pitch);

	hipMemcpy2D(h_A, pitch, devPtr, width* sizeof(float), height* sizeof(float), hipMemcpyDeviceToHost);

	hipFree(devPtr);


/*
	hipError_t hipMemcpy2D	(	void * 	dst, 
size_t 	dpitch,
const void * 	src,
size_t 	spitch,
size_t 	width,
size_t 	height,
enum hipMemcpyKind 	kind	 
)	

dst 	- Destination memory address
dpitch 	- Pitch of destination memory
src 	- Source memory address
spitch 	- Pitch of source memory
width 	- Width of matrix transfer (columns in bytes)
height 	- Height of matrix transfer (rows)
kind 	- Type of transfer



hipError_t hipMallocPitch	(	void ** 	devPtr,
size_t * 	pitch,
size_t 	width,
size_t 	height
)

devPtr 	- Pointer to allocated pitched device memory
pitch 	- Pitch for allocation
width 	- Requested pitched allocation width (in bytes)
height 	- Requested pitched allocation height


*/
 
