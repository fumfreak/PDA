#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>

#define dim 10;
#define RAND_MAX = 500;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *A, int *B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	for(int k = 0; k < dim; k++){
		_syncthreads();
		if(A[i+dim] + A[j+dim] < A[i+j])
			B[i+j] = A[i+dim] + A[j+dim];
	}
		
    
}

void init(int A){
	
	for(int i = 0; i < dim; i++)
		for(int j = 0; j < dim; j++)
			A[i][j] = rand();
}

void write(int A){

	for (int i = 0; i < dim; i++){
		for (int j = 0; j < dim; j++)
			printf("%d", A[i][j]);
		printf("/n");
	}
}


int main()
{
	int *d_A, *h_A, *d_B;
	int size = dim*dim;

	hipMalloc(&d_A, size);	
	hipMalloc(&d_B, size);
	
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	dim3 threadBlocks (n, n);

	addKernel <<<1, threadBlocks>>> (d_A);

	hipMemcpy(h_A, d_B, size, hipMemcpyDeviceToHost);
	
	write(h_A);

	hipFree(d_A);
	hipFree(d_B);
	
	return cudaStatus;
}

